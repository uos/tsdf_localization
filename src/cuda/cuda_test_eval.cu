#include "hip/hip_runtime.h"
#include <iostream>

//#include <boost/filesystem.hpp>

#include <tsdf_localization/cuda/cuda_evaluator.h>
#include <tsdf_localization/util/mcl_file.h>
#include <tsdf_localization/map/map_util.h>

#include <tsdf_localization/cuda/cuda_sub_voxel_map.h>

using namespace tsdf_localization;

int main(int argc, char** argv)
{
    if (argc != 3)
    {
        std::cout << "usage: " << argv[0] << " <mcl-file> <map-file>" << std::endl;
        return 0;
    }

    std::string mcl_name(argv[1]);
    std::string map_name(argv[2]);

    std::cout << "Read mcl data from file..." << std::endl;

    MCLFile file(mcl_name);

    std::vector<CudaPoint> points;
    std::vector<int> rings;
    std::vector<Particle> particles;
    std::array<FLOAT_T, 16> tf_matrix;
   
    FLOAT_T x, y, z, q_1, q_2, q_3, q_4;

    file.read(points, rings, particles, tf_matrix, x, y, z, q_1, q_2, q_3, q_4);

    std::unordered_set<SortClass, hash> point_set;

    for (auto index = 0u; index < points.size(); ++index)
    {
        CudaPoint center = {static_cast<float>(std::floor(points[index].x / 0.064) * 0.064 + 0.032), 
                            static_cast<float>(std::floor(points[index].y / 0.064) * 0.064 + 0.032), 
                            static_cast<float>(std::floor(points[index].z / 0.064) * 0.064 + 0.032)};
        
        point_set.insert(SortClass(rings[index], index, center));
    }

    std::vector<SortClass> reduced_points;
    reduced_points.resize(point_set.size());
    std::copy(point_set.begin(), point_set.end(), reduced_points.begin());

    std::vector<CudaPoint> ordered_points;
    ordered_points.reserve(reduced_points.size());

    std::vector<std::vector<std::pair<int, CudaPoint>>> sort_points(16);

    struct
    {
        bool operator()(std::pair<int, CudaPoint>& a, std::pair<int, CudaPoint>& b)
        {
            return a.first < b.first;
        }
    } customComp;


    for (auto& curr_ring : sort_points)
    {
        curr_ring.reserve(1500);
    }

    for (const auto& point : reduced_points)
    {
        sort_points[point.ring_].push_back(std::make_pair(point.index_, point.point_));
    }

    for (auto& curr_ring : sort_points)
    {
        std::sort(curr_ring.begin(), curr_ring.end(), customComp);
    }

    for (auto index = 0u; index < sort_points.size(); ++index)
    {
        auto& curr_ring = sort_points[index];

        for (auto& point : curr_ring)
        {
            ordered_points.push_back(point.second);
        }
    }

    std::cout << "Original cloud size: " << ordered_points.size() << std::endl;
    std::cout << "Reduced cloud size: " << reduced_points.size() << std::endl;
    std::cout << "Reduction ratio: " << static_cast<double>(reduced_points.size()) / ordered_points.size() * 100 << "%" << std::endl;

    std::cout << "Create map from file..." << std::endl;

    std::vector<CudaPoint> free_map;
    auto map = createTSDFMap<CudaSubVoxelMap<FLOAT_T, FLOAT_T>, FLOAT_T, FLOAT_T>(map_name, free_map);

    std::cout << "\nStart evaluation of the cuda kernel..." << std::endl;

    std::cout << "Generate device context..." << std::endl;
    
    CudaEvaluator evaluator(*map, false);

    std::cout << "Execute kernel..." << std::endl;

    geometry_msgs::Pose center_pose;
    center_pose.position.x = x;
    center_pose.position.y = y;
    center_pose.position.z = z;

    center_pose.orientation.w = q_1;
    center_pose.orientation.x = q_2;
    center_pose.orientation.y = q_3;
    center_pose.orientation.z = q_4;

    evaluator.evaluate(particles, ordered_points, tf_matrix.data());

    std::cout << "\nEvaluation finished!" << std::endl;

    return 0;
}
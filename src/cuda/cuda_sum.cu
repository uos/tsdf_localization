#include "hip/hip_runtime.h"
#include <cuda/cuda_sum.h>

#include <cmath>

namespace mcl
{

__device__ void warpReduce(volatile FLOAT_T* sdata, unsigned int tid)
{
    if(blockDim.x >= 64) sdata[tid] += sdata[tid + 32];
    if(blockDim.x >= 32) sdata[tid] += sdata[tid + 16];
    if(blockDim.x >= 16) sdata[tid] += sdata[tid + 8];
    if(blockDim.x >=  8) sdata[tid] += sdata[tid + 4];
    if(blockDim.x >=  4) sdata[tid] += sdata[tid + 2];
    if(blockDim.x >=  2) sdata[tid] += sdata[tid + 1];
}

__global__ void chunk_sums_kernel(const FLOAT_T* data, unsigned int data_size, unsigned int chunkSize, FLOAT_T* res)
{

    __shared__ FLOAT_T sdata[1024];
    
    const unsigned int tid = threadIdx.x;
    const unsigned int globId = chunkSize * blockIdx.x + threadIdx.x;
    const unsigned int rows = (chunkSize + blockDim.x - 1) / blockDim.x;


    sdata[tid] = 0.0;
    for(unsigned int i=0; i<rows; i++)
    {
        if(tid + blockDim.x * i < chunkSize)
        {
            auto calc = globId + blockDim.x * i;

            if (calc < data_size)
            {
                sdata[threadIdx.x] += data[calc];
            }
        }
    }
    __syncthreads();

    for(unsigned int s= blockDim.x / 2; s > 32; s >>= 1)
    {
        if(tid < s)
        {
            sdata[tid] += sdata[tid + s];
        }
        __syncthreads();
    }

    if(tid < blockDim.x / 2 && tid < 32)
    {
        warpReduce(sdata, tid);
    }

    if(tid == 0)
    {
        res[blockIdx.x] = sdata[0];
    }
}

void chunk_sums(const FLOAT_T* data_d, unsigned int data_size, FLOAT_T* res_d, unsigned int Nchunks, unsigned int chunkSize)
{
    if(chunkSize >= 1024) 
    {
        chunk_sums_kernel<<<Nchunks, 1024>>>(data_d, data_size, chunkSize, res_d);
    } 
    else if(chunkSize >= 512) 
    {
        chunk_sums_kernel<<<Nchunks, 512>>>(data_d, data_size, chunkSize, res_d);
    } 
    else if(chunkSize >= 256) 
    {
        chunk_sums_kernel<<<Nchunks, 256>>>(data_d, data_size, chunkSize, res_d);
    } 
    else if(chunkSize >= 128) 
    {
        chunk_sums_kernel<<<Nchunks, 128>>>(data_d, data_size, chunkSize, res_d);
    } 
    else if(chunkSize >= 64) 
    {
        chunk_sums_kernel<<<Nchunks, 64>>>(data_d, data_size, chunkSize, res_d);
    } 
    else if(chunkSize >= 32) 
    {
        chunk_sums_kernel<<<Nchunks, 32>>>(data_d, data_size, chunkSize, res_d);
    } 
    else if(chunkSize >= 16) 
    {
        chunk_sums_kernel<<<Nchunks, 16>>>(data_d, data_size, chunkSize, res_d);
    } 
    else if(chunkSize >= 8) 
    {
        chunk_sums_kernel<<<Nchunks, 8>>>(data_d, data_size, chunkSize, res_d);
    } 
    else if(chunkSize >= 4) 
    {
        chunk_sums_kernel<<<Nchunks, 4>>>(data_d, data_size, chunkSize, res_d);
    } 
    else if(chunkSize >= 2) 
    {
        chunk_sums_kernel<<<Nchunks, 2>>>(data_d, data_size, chunkSize, res_d);
    } 
    else if(chunkSize >= 1) 
    {
        chunk_sums_kernel<<<Nchunks, 1>>>(data_d, data_size, chunkSize, res_d);
    }
}

FLOAT_T* sumBatched(const FLOAT_T* data, size_t data_size, size_t batchSize)
{
    size_t Nchunks = std::ceil(data_size / static_cast<double>(batchSize));

    FLOAT_T* sums;
    hipMalloc((FLOAT_T**) &sums, sizeof(FLOAT_T) * Nchunks);
    chunk_sums(data, data_size, sums, Nchunks, batchSize);

    return sums;
}

double logN(double base, double x) 
{
    return log(x) / log(base);
}

FLOAT_T* sumBatchedRecursive(const FLOAT_T* data, size_t data_size)
{
    if (data_size <= 1024)
    {
        return sumBatched(data, data_size, data_size);
    }

    size_t Nchunks = std::ceil(data_size / static_cast<double>(1024));

    FLOAT_T* result = sumBatched(data, data_size, 1024);
    hipDeviceSynchronize();
    size_t remain = std::ceil(data_size / static_cast<double>(1024));

    FLOAT_T* tmp_data;
    hipMalloc((FLOAT_T**) &tmp_data, remain * sizeof(FLOAT_T));

    while (remain > 1)
    {
        Nchunks = std::ceil(remain / static_cast<double>(1024));

        chunk_sums(result, remain, tmp_data, Nchunks, 1024);
        hipDeviceSynchronize();
        remain = std::ceil(remain / static_cast<double>(1024));

        auto swap = tmp_data;
        tmp_data = result;
        result = swap;
    }

    hipFree(tmp_data);

    return result;
}

FLOAT_T weightSum(const FLOAT_T* data, size_t data_size)
{
    FLOAT_T weight_sum = 0;

    auto d_sums = sumBatchedRecursive(data, data_size);
    cudaCheck();

    hipMemcpy(&weight_sum, d_sums, sizeof(FLOAT_T), hipMemcpyDeviceToHost);

    hipFree(d_sums);

    return weight_sum;
}


}   // namespace mcl

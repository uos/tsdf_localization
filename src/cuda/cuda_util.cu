#include <tsdf_localization/cuda/cuda_util.h>

#ifdef __HIPCC__

#include <iostream>
#include <stdexcept>

void cudaCheck()
{
    auto code = hipPeekAtLastError();

    if (code != hipSuccess)
    {
        std::cout << "CUDA error: " << hipGetErrorString(code) << std::endl;
        throw std::runtime_error("CUDA error occured!");
    }
}

#endif
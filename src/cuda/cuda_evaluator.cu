#include "hip/hip_runtime.h"
#include <tsdf_localization/cuda/cuda_evaluator.h>
#include <stdio.h>

#include <tsdf_localization/cuda/cuda_util.h>

// #include <sensor_msgs/point_cloud2_iterator.hpp>

// #include <tf2/LinearMath/Quaternion.h>
// #include <tf2_geometry_msgs/tf2_geometry_msgs.hpp>

#include <tsdf_localization/util/runtime_evaluator.h>
#include <map>

#include <tsdf_localization/cuda/cuda_data.h>
#include <tsdf_localization/cuda/cuda_eval_particles.h>
#include <tsdf_localization/cuda/cuda_sum.h>

namespace tsdf_localization 
{

CudaEvaluator::CudaEvaluator(CudaSubVoxelMap<FLOAT_T, FLOAT_T>& map, bool per_point, FLOAT_T a_hit, FLOAT_T a_range, FLOAT_T a_max, FLOAT_T max_range) : 
d_map_(nullptr), per_point_(per_point), d_grid_occ_(nullptr), d_data_(nullptr), d_particles_(nullptr), d_particles_ordered_(nullptr), particles_reserved_(0), d_points_(nullptr), d_points_ordered_(nullptr), points_reserved_(0), d_transform_(nullptr), d_new_weights_(nullptr), d_point_weights_(nullptr), point_weights_size_(0),
p_x_{nullptr}, p_y_{nullptr}, p_z_{nullptr}, sin_a_{nullptr}, cos_a_{nullptr}, sin_b_{nullptr}, cos_b_{nullptr}, sin_c_{nullptr}, cos_c_{nullptr}/*, dev_random_{nullptr}*/,
a_hit_{a_hit}, a_range_(a_range), a_max_(a_max), 
max_range_(max_range), inv_max_range_(1.0 / max_range), max_range_squared_(max_range * max_range)
{
  try
  {
    hipMemcpyToSymbol(HIP_SYMBOL(const_map_coef_), &(map.coef()), sizeof(CudaSubVoxelMap<FLOAT_T, FLOAT_T>::MapCoef));
    hipMemcpyToSymbol(HIP_SYMBOL(const_a_hit_), &a_hit_, sizeof(FLOAT_T));
    hipMemcpyToSymbol(HIP_SYMBOL(const_a_range_), &a_range_, sizeof(FLOAT_T));
    hipMemcpyToSymbol(HIP_SYMBOL(const_a_max_), &a_max_, sizeof(FLOAT_T));

    hipMemcpyToSymbol(HIP_SYMBOL(const_max_range_), &max_range_, sizeof(FLOAT_T));
    hipMemcpyToSymbol(HIP_SYMBOL(const_max_range_squared_), &max_range_squared_, sizeof(FLOAT_T));
    hipMemcpyToSymbol(HIP_SYMBOL(const_inv_max_range_), &inv_max_range_, sizeof(FLOAT_T));


    cudaCheck();

    hipMalloc((OCC_T**) &(g_grid_occ_), map.gridOccBytes());
    hipMemcpy(g_grid_occ_, map.rawGridOcc(), map.gridOccBytes(), hipMemcpyHostToDevice);

    hipMalloc((FLOAT_T**) &(g_data_), map.dataBytes());
    hipMemcpy(g_data_, map.rawData(), map.dataBytes(), hipMemcpyHostToDevice);
    
    cudaCheck();

    hipMalloc((FLOAT_T**) &(d_transform_), TRANSFORM_BYTES);

    cudaCheck();
  }
  catch (std::runtime_error&)
  {
    throw std::runtime_error("Error while creating the CUDA context for the map!");
  }

  hipFuncSetCacheConfig(reinterpret_cast<const void*>(cudaEvaluateParticlesOrdered), hipFuncCachePreferL1);
}

struct lower_than_key
{
    inline bool operator()(const Particle& particle1, const Particle& particle2) 
    {
        return particle1.first[0] < particle2.first[0];
    }
};

inline FLOAT_T particle_dist(const Particle& particle1, const Particle& particle2)
{
    auto dx = particle1.first[0] - particle2.first[0];
    auto dy = particle1.first[1] - particle2.first[1];
    auto dz = particle1.first[2] - particle2.first[2];

    return sqrt((dx * dx + dy * dy + dz * dz));
} 

// geometry_msgs::msg::PoseWithCovariance CudaEvaluator::evaluate(std::vector<Particle>& particles, const sensor_msgs::msg::PointCloud2& real_cloud, FLOAT_T tf_matrix[16])
// {
//   sensor_msgs::PointCloud2ConstIterator<float> iter_x(real_cloud, "x");
//   sensor_msgs::PointCloud2ConstIterator<int> iter_ring(real_cloud, "ring");

//   std::vector<CudaPoint> points;
//   points.reserve(real_cloud.width);
  
//   std::multimap<int, CudaPoint> map;

//   for (; iter_x != iter_x.end(); ++iter_x)
//   {
//       map.insert(std::pair<int, CudaPoint>(iter_ring[0], {iter_x[0], iter_x[1], iter_x[2]}));

//       ++iter_ring;
//   }

//   for (const auto& entry : map)
//   {
//     points.push_back(entry.second);
//   }

//   std::unordered_set<CudaPoint, hash> point_set;

//   for (const auto& point : points)
//   {
//       CudaPoint center = {static_cast<float>(std::floor(point.x / 0.064) * 0.064 + 0.032), 
//                           static_cast<float>(std::floor(point.y / 0.064) * 0.064 + 0.032), 
//                           static_cast<float>(std::floor(point.z / 0.064) * 0.064 + 0.032)};
      
//       point_set.insert(center);
//   }

//   std::vector<CudaPoint> reduced_points;
//   reduced_points.resize(point_set.size());
//   std::copy(point_set.begin(), point_set.end(), reduced_points.begin());

//   return evaluate(particles, reduced_points, tf_matrix);
// }

Particle CudaEvaluator::evaluate(std::vector<Particle>& particles, const std::vector<CudaPoint>& points, FLOAT_T tf_matrix[16])
{
  static auto& eval = RuntimeEvaluator::get_instance();

  if (points.size() == 0)
  {
    return Particle();
  }

  eval.start("init_kernel");

  try
  {
    auto particle_bytes = sizeof(Particle) * particles.size();
    
    std::vector<FLOAT_T> particles_ordered(6 * particles.size());
    auto particle_ordered_bytes = sizeof(FLOAT_T) * particles_ordered.size();

    if(particles_reserved_ < particles.size())
    {
      if (d_particles_ != nullptr)
      {
        hipFree(d_particles_);
        hipFree(d_particles_ordered_);

        hipFree(p_x_);
        hipFree(p_y_);
        hipFree(p_z_);

        hipFree(sin_a_);
        hipFree(cos_a_);

        hipFree(sin_b_);
        hipFree(cos_b_);

        hipFree(sin_c_);
        hipFree(cos_c_);

        if (!per_point_)
        {
          hipFree(d_new_weights_);
        }

        d_particles_ = nullptr;
        d_particles_ordered_ = nullptr;
        d_new_weights_ = nullptr;

        p_x_ = nullptr;
        p_y_ = nullptr;
        p_z_ = nullptr;

        sin_a_ = nullptr;
        cos_a_ = nullptr;

        sin_b_ = nullptr;
        cos_b_ = nullptr;

        sin_c_ = nullptr;
        cos_c_ = nullptr;
      }

      cudaCheck();

      hipMalloc((Particle**) &d_particles_, particle_bytes);
      hipMalloc((FLOAT_T**) &d_particles_ordered_, particle_ordered_bytes);

      hipMalloc((FLOAT_T**) &p_x_, sizeof(FLOAT_T) * particles.size());
      hipMalloc((FLOAT_T**) &p_y_, sizeof(FLOAT_T) * particles.size());
      hipMalloc((FLOAT_T**) &p_z_, sizeof(FLOAT_T) * particles.size());

      hipMalloc((FLOAT_T**) &sin_a_, sizeof(FLOAT_T) * particles.size());
      hipMalloc((FLOAT_T**) &cos_a_, sizeof(FLOAT_T) * particles.size());

      hipMalloc((FLOAT_T**) &sin_b_, sizeof(FLOAT_T) * particles.size());
      hipMalloc((FLOAT_T**) &cos_b_, sizeof(FLOAT_T) * particles.size());

      hipMalloc((FLOAT_T**) &sin_c_, sizeof(FLOAT_T) * particles.size());
      hipMalloc((FLOAT_T**) &cos_c_, sizeof(FLOAT_T) * particles.size());
      
      cudaCheck();

      if (!per_point_)
      {
        hipMalloc((FLOAT_T**) &d_new_weights_, sizeof(FLOAT_T) * particles.size());
      }

      cudaCheck();

      new_weights_.resize(particles.size());

      particles_reserved_ = particles.size();
    }

    for (auto index = 0u; index < particles.size(); ++index)
    {
      const auto& particle = particles[index];

      particles_ordered[                       index] = particle.first[0];
      particles_ordered[    particles.size() + index] = particle.first[1];
      particles_ordered[2 * particles.size() + index] = particle.first[2];
      particles_ordered[3 * particles.size() + index] = particle.first[3];
      particles_ordered[4 * particles.size() + index] = particle.first[4];
      particles_ordered[5 * particles.size() + index] = particle.first[5];
    }

    hipMemcpy(d_particles_, particles.data(), particle_bytes, hipMemcpyHostToDevice);
    hipMemcpy(d_particles_ordered_, particles_ordered.data(), particle_ordered_bytes, hipMemcpyHostToDevice);
    cudaCheck();
  }
  catch (std::runtime_error&)
  {
    throw std::runtime_error("Error while preparing the particles for thhe CUDA evaluation kernel!");
  }

  try
  {
    auto point_bytes = sizeof(CudaPoint) * points.size();

    std::vector<FLOAT_T> h_points_ordered(points.size() * 3);
    auto points_ordered_bytes = h_points_ordered.size() * sizeof(FLOAT_T);

    if(points_reserved_ < points.size())
    {
      if (per_point_)
      {
        if (d_points_ordered_ != nullptr)
        {
          hipFree(d_points_ordered_);
        }

        hipMalloc((FLOAT_T**) &d_points_ordered_, points_ordered_bytes);
      }
      //else
      {
        if (d_points_ != nullptr)
        {
          hipFree(d_points_);
        }

        hipMalloc((CudaPoint**) &d_points_, point_bytes);
      }

      points_reserved_ = points.size();
    }

    hipMemcpy(d_points_, points.data(), point_bytes, hipMemcpyHostToDevice);
    cudaCheck();

    if (per_point_)
    {
      for (auto index = 0u; index < points.size(); ++index)
      {
        const auto& point = points[index];

        h_points_ordered[                    index] = point.x;
        h_points_ordered[    points.size() + index] = point.y;
        h_points_ordered[2 * points.size() + index] = point.z;
      }

      hipMemcpy(d_points_ordered_, h_points_ordered.data(), points_ordered_bytes, hipMemcpyHostToDevice);
      cudaCheck();
    }
  }
  catch (std::runtime_error&)
  {
    throw std::runtime_error("Error while preparing the cloud points for the CUDA evaluation kernel!");
  }

  try
  {
    hipMemcpy(d_transform_, tf_matrix, TRANSFORM_BYTES, hipMemcpyHostToDevice);
    hipMemcpyToSymbol(HIP_SYMBOL(const_tf_matrix), tf_matrix, TRANSFORM_BYTES);
    cudaCheck();
  }
  catch (std::runtime_error&)
  {
    throw std::runtime_error("Error while preparing the cloud points for the CUDA evaluation kernel!");
  }

  eval.stop("init_kernel");

  eval.start("exec_kernel", true);

  int blocksize = 64;

  try
  {

    if (per_point_)
    {
      auto num_data_points = particles.size() * points.size();
      auto num_data_points_bytes = num_data_points * sizeof(FLOAT_T);

      dim3 block(blocksize, 1);
      dim3 grid((num_data_points + block.x - 1) / block.x, 1);

      if (point_weights_size_ < num_data_points_bytes)
      {
        if (d_point_weights_ != nullptr)
        {
          hipFree(d_point_weights_);
          d_point_weights_ = nullptr;
          cudaCheck();
        }

        hipMalloc((FLOAT_T**) &d_point_weights_, num_data_points_bytes);
        cudaCheck();
        point_weights_size_ = num_data_points_bytes;
      }

      cudaEvaluatePointsOrdered<<<grid, block>>>(g_grid_occ_, g_data_, g_map_coef_, d_points_ordered_, points.size(), d_particles_ordered_, particles.size(), d_transform_, d_point_weights_);
      hipDeviceSynchronize();
      cudaCheck();

      if (d_new_weights_ != nullptr)
      {
        hipFree(d_new_weights_);
        d_new_weights_ = nullptr;
        cudaCheck();
      }

      d_new_weights_ = sumBatched(d_point_weights_, num_data_points, points.size());
      cudaCheck();

      //hipFree(d_point_weights);
    }
    else
    {
      dim3 block(blocksize, 1);
      dim3 grid((particles.size() + block.x - 1) / block.x, 1);

      cudaEvaluateParticlesOrdered<<<grid, block>>>(g_grid_occ_, g_data_, g_map_coef_, d_points_, points.size(), d_particles_ordered_, particles.size(), d_transform_, d_new_weights_);
    }
    
    hipDeviceSynchronize();
    cudaCheck();
  }
  catch (std::runtime_error&)
  {
    throw std::runtime_error("Error during the execution of the evaluation kernel occured!");
  }

  eval.stop("exec_kernel");

  eval.start("weight_update");
  
  FLOAT_T weight_sum = weightSum(d_new_weights_, particles.size());

  if(weight_sum == 0.0)
  {
    throw std::runtime_error("No particle is valid!");
  }

  Particle average_particle;
  // geometry_msgs::msg::PoseWithCovariance average_pose;

  // FLOAT_T variance_x = 0.0;
  // FLOAT_T variance_y = 0.0;
  // FLOAT_T variance_z = 0.0;

  // FLOAT_T variance_roll = 0.0;
  // FLOAT_T variance_pitch = 0.0;
  // FLOAT_T variance_yaw = 0.0;

  dim3 block(blocksize, 1);
  dim3 grid((particles.size() + block.x - 1) / block.x, 1);
  weight_particles<<<grid, block>>>(d_particles_ordered_, particles.size(), d_new_weights_, weight_sum, p_x_, p_y_, p_z_, sin_a_, cos_a_,  sin_b_, cos_b_,  sin_c_, cos_c_);
  hipDeviceSynchronize();

  average_particle.first[0] = weightSum(p_x_, particles.size());
  average_particle.first[1] = weightSum(p_y_, particles.size());
  average_particle.first[2] = weightSum(p_z_, particles.size());
  average_particle.first[3] = atan2(weightSum(sin_a_, particles.size()), weightSum(cos_a_, particles.size()));
  average_particle.first[4] = atan2(weightSum(sin_b_, particles.size()), weightSum(cos_b_, particles.size()));
  average_particle.first[5] = atan2(weightSum(sin_c_, particles.size()), weightSum(cos_c_, particles.size()));


  try
  {
    hipMemcpy(new_weights_.data(), d_new_weights_, sizeof(FLOAT_T) * particles.size(), hipMemcpyDeviceToHost);
    cudaCheck();
  }
  catch (std::runtime_error&)
  {
    throw std::runtime_error("Error occured while copying data back from the gpu!");
  }

  for(auto index = 0u; index < particles.size(); index++)
  {
    particles[index].second = new_weights_[index];
  }

  // average_pose.pose.position.x = average_particle.first[0];
  // average_pose.pose.position.y = average_particle.first[1];
  // average_pose.pose.position.z = average_particle.first[2];

  // tf2::Quaternion tf_quaternion;
  // tf_quaternion.setRPY(average_particle.first[3] , average_particle.first[4], average_particle.first[5]);
  // tf2::convert(tf_quaternion, average_pose.pose.orientation);

  // average_pose.covariance.data()[0] = variance_x;
  // average_pose.covariance.data()[7] = variance_y;
  // average_pose.covariance.data()[14] = variance_z;
  // average_pose.covariance.data()[21] = variance_roll;
  // average_pose.covariance.data()[28] = variance_pitch;
  // average_pose.covariance.data()[35] = variance_yaw;

  eval.stop("weight_update");

  return average_particle;
}

CudaEvaluator::~CudaEvaluator()
{
  if (p_x_ != nullptr)
  {
    hipFree(p_x_);
    cudaCheck();
  }

  if (p_y_ != nullptr)
  {
    hipFree(p_y_);
    cudaCheck();
  }

  if (p_z_ != nullptr)
  {
    hipFree(p_z_);
    cudaCheck();
  }

  if (d_grid_occ_ != nullptr)
  {
    hipFree(d_grid_occ_);
    cudaCheck();
  }

  if (d_data_ != nullptr)
  {
    hipFree(d_data_);
    cudaCheck();
  }

  if (d_map_ != nullptr)
  {
    hipFree(d_map_);
    cudaCheck();
  }

  if (d_particles_ != nullptr)
  {
    hipFree(d_particles_);
    cudaCheck();
  }

  if (d_particles_ordered_ != nullptr)
  {
    hipFree(d_particles_ordered_);
    cudaCheck();
  }

  if (d_points_ != nullptr)
  {
    hipFree(d_points_);
    cudaCheck();
  }

  if (d_points_ordered_ != nullptr)
  {
    hipFree(d_points_ordered_);
    cudaCheck();
  }

  if (d_transform_ != nullptr)
  {
    hipFree(d_transform_);
    cudaCheck();
  }

  if (d_new_weights_ != nullptr)
  {
    hipFree(d_new_weights_);
    cudaCheck();
  }

  if (g_grid_occ_ != nullptr)
  {
    hipFree(g_grid_occ_);
    cudaCheck();
  }

  if (g_data_ != nullptr)
  {
    hipFree(g_data_);
    cudaCheck();
  }

  if (g_map_coef_ != nullptr)
  {
    hipFree(g_map_coef_);
    cudaCheck();
  }

  if (d_tex_grid_ != nullptr)
  {
    hipFree(d_tex_grid_);
    cudaCheck();
  }

  if (d_point_weights_ != nullptr)
  {
    hipFree(d_point_weights_);
    cudaCheck();
  }

  if (sin_a_ != nullptr)
  {
    hipFree(sin_a_);
    cudaCheck();
  }

  if (cos_a_ != nullptr)
  {
    hipFree(cos_a_);
    cudaCheck();
  }

  if (sin_b_ != nullptr)
  {
    hipFree(sin_b_);
    cudaCheck();
  }

  if (cos_b_ != nullptr)
  {
    hipFree(cos_b_);
    cudaCheck();
  }

  if (sin_c_ != nullptr)
  {
    hipFree(sin_c_);
    cudaCheck();
  }

  if (cos_c_ != nullptr)
  {
    hipFree(cos_c_);
    cudaCheck();
  }
}

} // namespace tsdf_localization
